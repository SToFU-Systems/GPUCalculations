﻿/*
 *
 *  Copyright (C) 2023, SToFU Systems S.L.
 *  All rights reserved.
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License along
 *  with this program; if not, write to the Free Software Foundation, Inc.,
 *  51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 *
 */
#include "hip/hip_runtime.h"
#include ""

#include "tests.cuh"

inline uint64_t getTime()
{
    return std::chrono::time_point_cast<std::chrono::milliseconds>(std::chrono::high_resolution_clock::now()).time_since_epoch().count();
}

void crc32Ex(size_t count, size_t size)
{
    std::vector<std::vector<unsigned char>> buffers = tests::generateBuffers(count, size);

    auto startGPU = getTime(); // Start time for GPU function
    auto resultGPU = tests::testCRC32GPU(buffers); // Call the GPU function
    std::cout << "GPU crc32 Execution Time: " << getTime() - startGPU << " milliseconds" << std::endl; // Execution time of GPU function in milliseconds

    auto startCPU = getTime(); // Start time for CPU function
    auto resultCPU = tests::testCRC32CPU(buffers); // Call the CPU function
    std::cout << "CPU crc32 Execution Time: " << getTime() - startCPU << " milliseconds" << std::endl; // Execution time of CPU function in milliseconds
}

void sha512Ex(size_t count, size_t size)
{
    std::vector<std::vector<unsigned char>> buffers = tests::generateBuffers(count, size);

    auto startGPU = getTime(); // Start time for GPU function
    auto resultGPU = tests::testSHA512GPU(buffers); // Call the GPU function
    std::cout << "GPU sha512 Execution Time: " << getTime() - startGPU << " milliseconds" << std::endl; // Execution time of GPU function in milliseconds

    auto startCPU = getTime(); // Start time for CPU function
    auto resultCPU = tests::testSHA512CPU(buffers); // Call the CPU function
    std::cout << "CPU sha512 Execution Time: " << getTime() - startCPU << " milliseconds" << std::endl; // Execution time of CPU function in milliseconds
}

int main()
{
    crc32Ex(1000, 1000);
    sha512Ex(1000, 1000);
 
    return 0;
}

