#include "hip/hip_runtime.h"
/*
 *
 *  Copyright (C) 2023, SToFU Systems S.L.
 *  All rights reserved.
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License along
 *  with this program; if not, write to the Free Software Foundation, Inc.,
 *  51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 *
 */
#include "crc32.cuh"

namespace crc32
{
    /*
    * FUNCTION: __device__ __host__ uint32_t crc32
    *
    * ARGS:
    * const uint8_t* buffer - Input buffer containing data for CRC32 calculation.
    * int size - Size of the input buffer.
    *
    * DESCRIPTION:
    * This function calculates the CRC32 checksum for a given input buffer on both CPU and GPU devices.
    * The CRC32 calculation algorithm used is the standard CRC32 polynomial with initial value of 0xFFFFFFFF and final XOR of 0xFFFFFFFF.
    * The function iterates through each byte in the input buffer using a for loop, performing bitwise XOR and shift operations to calculate the CRC32 value.
    * The calculated CRC32 value is then bitwise negated (~crc) and returned as the final result.
    *
    * RETURN VALUE:
    * uint32_t - The calculated CRC32 checksum for the input buffer.
    * This function returns a 32-bit unsigned integer representing the CRC32 checksum value.
    */
    __device__ __host__ uint32_t crc32(const uint8_t* buffer, int size)
    {
        uint32_t crc = 0xFFFFFFFF;
        for (int i = 0; i < size; ++i)
        {
            crc ^= buffer[i];
            for (int j = 0; j < 8; ++j)
                crc = (crc >> 1) ^ (0xEDB88320 & (-(crc & 1)));
        }
        return ~crc;
    }

    /*
    * FUNCTION: __global__ void crc32Kernel
    *
    * ARGS:
    * In uint8_t* buffers - Input buffer containing data for CRC32 calculation.
    * Out uint32_t* crcResults - Output buffer to store CRC32 results.
    * int numBuffers - Number of input buffers.
    * int bufferSize - Size of each input buffer.
    *
    * DESCRIPTION:
    * This is a CUDA kernel function for calculating CRC32 checksums in parallel on a GPU device.
    * Each thread in the GPU grid corresponds to a unique thread identifier (tid) calculated from blockIdx.x and blockDim.x.
    * The bufferIndex is calculated based on tid and bufferSize to determine the starting index of the current buffer to be processed.
    * The function performs CRC32 calculation on each buffer by iterating through each byte in the buffer using a for loop.
    * The calculated CRC32 value is then saved to the crcResults array at the corresponding tid index.
    *
    * RETURN VALUE: void
    * This function does not return a value.
    */
    __global__ void crc32Kernel(_In_ const uint8_t* buffers, _Out_ uint32_t* crcResults, int numBuffers, int bufferSize)
    {
        /* Calculate unique thread identifier */
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        /* Calculate index of the current buffer */
        int bufferIndex = tid * bufferSize;

        /* Check if buffer index is within valid range */
        if (bufferIndex < numBuffers * bufferSize)
            /* Call crc32 function to calculate CRC32 for the current buffer */
            crcResults[tid] = crc32(buffers + bufferIndex, bufferSize);
    }
}